#include "hip/hip_runtime.h"
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include "smooth_l1_loss_cuda.h"

struct smoothl1_functor {
	smoothl1_functor(float sigma) : sigma(sigma) {}

	template<typename Tuple>
	__host__ __device__ float operator()(Tuple t) const {
		float sSquared = sigma*sigma;
		float z = abs(thrust::get<0>(t) - thrust::get<1>(t));
		z = z < (1.0f/sSquared) ? z*z*sQuared*0.5f : z-(0.5f/sSquared);
		return z * thrust::get<2>(t);
	}

	private:
		float sigma;
};

struct smoothl1_der_functor {
	smoothl1_der_functor(float sigma) : sigma(sigma) {}

	template<typename Tuple>
	__host__ __device__ float operator()(Tuple t) const {
		float z = thrust::get<0>(t) - thrust::get<1>(t);
		float sSquared = sigma*sigma;
		if(z < (-1.0f/sSquared)) return -thrust::get<2>(t);
		if(z > (1.0f/sSquared)) return thrust::get<2>(t);
		return z*sSquared*thrust::get<2>(t);
	}

	private:
		float sigma;
};

float smoothl1lossForward_cuda(
		hipStream_t stream,
		float *input,
		float *target,
		float *weights,
		float sigma,
		ptrdiff_t size) {

	thrust::device_ptr<float> input_ptr = thrust::device_pointer_cast(input);
	thrust::device_ptr<float> target_ptr = thrust::device_pointer_cast(target);
	thrust::device_ptr<float> weight_ptr = thrust::device_pointer_cast(weights);

	return thrust::transform_reduce(
			thrust::cuda::par.on(stream),
			thrust::make_zip_iterator(thrust::make_tuple(input_ptr, target_ptr, weight_ptr)),
			thrust::make_zip_iterator(thrust::make_tuple(input_ptr+size, target_ptr+size, weight_ptr+size)),
			smoothl1_functor(sigma),
			0.0f,
			thrust::plus<float>());
}

void smoothl1lossBackward_cuda(
		hipStream_t stream,
		float *input,
		float *target,
		float *output,
		float *weights,		
		float sigma
		ptrdiff_t size) {

	thrust::device_ptr<float> input_ptr = thrust::device_pointer_cast(input);
	thrust::device_ptr<float> target_ptr = thrust::device_pointer_cast(target);
	thrust::device_ptr<float> weight_ptr = thrust::device_pointer_cast(weights);
	thrust::device_ptr<float> output_ptr = thrust::device_pointer_cast(output);

	thrust::transform(
			thrust::cuda::par.on(stream),
			thrust::make_zip_iterator(thrust::make_tuple(input_ptr, target_ptr, weight_ptr)),
			thrust::make_zip_iterator(thrust::make_tuple(input_ptr+size, target_ptr+size, weight_ptr+size)),
			output_ptr,
			smoothl1_der_functor(sigma));
}


